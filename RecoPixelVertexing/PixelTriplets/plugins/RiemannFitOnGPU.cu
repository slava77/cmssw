#include "hip/hip_runtime.h"
//
// Author: Felice Pantaleo, CERN
//

#include <cstdint>

#include <hip/hip_runtime.h>

#include "CUDADataFormats/TrackingRecHit/interface/TrackingRecHit2DCUDA.h"
#include "FWCore/ServiceRegistry/interface/Service.h"
#include "HeterogeneousCore/CUDAServices/interface/CUDAService.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cuda_assert.h"
#include "RecoLocalTracker/SiPixelRecHits/interface/pixelCPEforGPU.h"
#include "RecoPixelVertexing/PixelTrackFitting/interface/RiemannFit.h"

#include "HelixFitOnGPU.h"

using HitsOnGPU = TrackingRecHit2DSOAView;
using Tuples = pixelTrack::HitContainer;
using OutputSoA = pixelTrack::TrackSoA;

using namespace Eigen;

template <int N>
__global__ void kernelFastFit(Tuples const *__restrict__ foundNtuplets,
                              CAConstants::TupleMultiplicity const *__restrict__ tupleMultiplicity,
                              uint32_t nHits,
                              HitsOnGPU const *__restrict__ hhp,
                              double *__restrict__ phits,
                              float *__restrict__ phits_ge,
                              double *__restrict__ pfast_fit,
                              uint32_t offset) {
  constexpr uint32_t hitsInFit = N;

  assert(hitsInFit <= nHits);

  assert(pfast_fit);
  assert(foundNtuplets);
  assert(tupleMultiplicity);

  // look in bin for this hit multiplicity
  auto local_start = (blockIdx.x * blockDim.x + threadIdx.x);

#ifdef RIEMANN_DEBUG
  if (0 == local_start)
    printf("%d Ntuple of size %d for %d hits to fit\n", tupleMultiplicity->size(nHits), nHits, hitsInFit);
#endif

  auto tuple_start = local_start + offset;
  if (tuple_start >= tupleMultiplicity->size(nHits))
    return;

  // get it from the ntuple container (one to one to helix)
  auto tkid = *(tupleMultiplicity->begin(nHits) + tuple_start);
  assert(tkid < foundNtuplets->nbins());

  assert(foundNtuplets->size(tkid) == nHits);

  Rfit::Map3xNd<N> hits(phits + local_start);
  Rfit::Map4d fast_fit(pfast_fit + local_start);
  Rfit::Map6xNf<N> hits_ge(phits_ge + local_start);

  // Prepare data structure
  auto const *hitId = foundNtuplets->begin(tkid);
  for (unsigned int i = 0; i < hitsInFit; ++i) {
    auto hit = hitId[i];
    // printf("Hit global: %f,%f,%f\n", hhp->xg_d[hit],hhp->yg_d[hit],hhp->zg_d[hit]);
    float ge[6];
    hhp->cpeParams().detParams(hhp->detectorIndex(hit)).frame.toGlobal(hhp->xerrLocal(hit), 0, hhp->yerrLocal(hit), ge);
    // printf("Error: %d: %f,%f,%f,%f,%f,%f\n",hhp->detInd_d[hit],ge[0],ge[1],ge[2],ge[3],ge[4],ge[5]);

    hits.col(i) << hhp->xGlobal(hit), hhp->yGlobal(hit), hhp->zGlobal(hit);
    hits_ge.col(i) << ge[0], ge[1], ge[2], ge[3], ge[4], ge[5];
  }
  Rfit::Fast_fit(hits, fast_fit);

  // no NaN here....
  assert(fast_fit(0) == fast_fit(0));
  assert(fast_fit(1) == fast_fit(1));
  assert(fast_fit(2) == fast_fit(2));
  assert(fast_fit(3) == fast_fit(3));
}

template <int N>
__global__ void kernelCircleFit(CAConstants::TupleMultiplicity const *__restrict__ tupleMultiplicity,
                                uint32_t nHits,
                                double B,
                                double *__restrict__ phits,
                                float *__restrict__ phits_ge,
                                double *__restrict__ pfast_fit_input,
                                Rfit::circle_fit *circle_fit,
                                uint32_t offset) {
  assert(circle_fit);
  assert(N <= nHits);

  // same as above...

  // look in bin for this hit multiplicity
  auto local_start = (blockIdx.x * blockDim.x + threadIdx.x);
  auto tuple_start = local_start + offset;
  if (tuple_start >= tupleMultiplicity->size(nHits))
    return;

  // get it for the ntuple container (one to one to helix)
  auto tkid = *(tupleMultiplicity->begin(nHits) + tuple_start);

  Rfit::Map3xNd<N> hits(phits + local_start);
  Rfit::Map4d fast_fit(pfast_fit_input + local_start);
  Rfit::Map6xNf<N> hits_ge(phits_ge + local_start);

  Rfit::VectorNd<N> rad = (hits.block(0, 0, 2, N).colwise().norm());

  Rfit::Matrix2Nd<N> hits_cov = Rfit::Matrix2Nd<N>::Zero();
  Rfit::loadCovariance2D(hits_ge, hits_cov);

  circle_fit[local_start] = Rfit::Circle_fit(hits.block(0, 0, 2, N), hits_cov, fast_fit, rad, B, true);

#ifdef RIEMANN_DEBUG
//  printf("kernelCircleFit circle.par(0,1,2): %d %f,%f,%f\n", tkid,
//         circle_fit[local_start].par(0), circle_fit[local_start].par(1), circle_fit[local_start].par(2));
#endif
}

template <int N>
__global__ void kernelLineFit(CAConstants::TupleMultiplicity const *__restrict__ tupleMultiplicity,
                              uint32_t nHits,
                              double B,
                              OutputSoA *results,
                              double *__restrict__ phits,
                              float *__restrict__ phits_ge,
                              double *__restrict__ pfast_fit_input,
                              Rfit::circle_fit *__restrict__ circle_fit,
                              uint32_t offset) {
  assert(results);
  assert(circle_fit);
  assert(N <= nHits);

  // same as above...

  // look in bin for this hit multiplicity
  auto local_start = (blockIdx.x * blockDim.x + threadIdx.x);
  auto tuple_start = local_start + offset;
  if (tuple_start >= tupleMultiplicity->size(nHits))
    return;

  // get it for the ntuple container (one to one to helix)
  auto tkid = *(tupleMultiplicity->begin(nHits) + tuple_start);

  Rfit::Map3xNd<N> hits(phits + local_start);
  Rfit::Map4d fast_fit(pfast_fit_input + local_start);
  Rfit::Map6xNf<N> hits_ge(phits_ge + local_start);

  auto const &line_fit = Rfit::Line_fit(hits, hits_ge, circle_fit[local_start], fast_fit, B, true);

  Rfit::fromCircleToPerigee(circle_fit[local_start]);

  results->stateAtBS.copyFromCircle(circle_fit[local_start].par,circle_fit[local_start].cov,
                                   line_fit.par,line_fit.cov,1.f/float(B),tkid);
  results->pt(tkid) =  B/std::abs(circle_fit[local_start].par(2));
  results->eta(tkid) =  asinhf(line_fit.par(0));
  results->chi2(tkid) = (circle_fit[local_start].chi2+line_fit.chi2)/(2*N-5);

#ifdef RIEMANN_DEBUG
  printf("kernelLineFit size %d for %d hits circle.par(0,1,2): %d %f,%f,%f\n",
         N,
         nHits,
         tkid,
         circle_fit[local_start].par(0),
         circle_fit[local_start].par(1),
         circle_fit[local_start].par(2));
  printf("kernelLineFit line.par(0,1): %d %f,%f\n", tkid, line_fit.par(0), line_fit.par(1));
  printf("kernelLineFit chi2 cov %f/%f %e,%e,%e,%e,%e\n",
         circle_fit[local_start].chi2,
         line_fit.chi2,
         circle_fit[local_start].cov(0, 0),
         circle_fit[local_start].cov(1, 1),
         circle_fit[local_start].cov(2, 2),
         line_fit.cov(0, 0),
         line_fit.cov(1, 1));
#endif
}

void HelixFitOnGPU::launchRiemannKernels(HitsOnCPU const &hh,
                                         uint32_t nhits,
                                         uint32_t maxNumberOfTuples,
                                         cuda::stream_t<> &stream) {
  assert(tuples_d);

  auto blockSize = 64;
  auto numberOfBlocks = (maxNumberOfConcurrentFits_ + blockSize - 1) / blockSize;

  //  Fit internals
  edm::Service<CUDAService> cs;
  auto hitsGPU_ = cs->make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix3xNd<4>) / sizeof(double), stream);
  auto hits_geGPU_ =
      cs->make_device_unique<float[]>(maxNumberOfConcurrentFits_ * sizeof(Rfit::Matrix6x4f) / sizeof(float), stream);
  auto fast_fit_resultsGPU_ =
      cs->make_device_unique<double[]>(maxNumberOfConcurrentFits_ * sizeof(Rfit::Vector4d) / sizeof(double), stream);
  auto circle_fit_resultsGPU_holder =
      cs->make_device_unique<char[]>(maxNumberOfConcurrentFits_ * sizeof(Rfit::circle_fit), stream);
  Rfit::circle_fit *circle_fit_resultsGPU_ = (Rfit::circle_fit *)(circle_fit_resultsGPU_holder.get());

  for (uint32_t offset = 0; offset < maxNumberOfTuples; offset += maxNumberOfConcurrentFits_) {
    // triplets
    kernelFastFit<3><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tuples_d,
                                                                    tupleMultiplicity_d,
                                                                    3,
                                                                    hh.view(),
                                                                    hitsGPU_.get(),
                                                                    hits_geGPU_.get(),
                                                                    fast_fit_resultsGPU_.get(),
                                                                    offset);
    cudaCheck(hipGetLastError());

    kernelCircleFit<3><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                      3,
                                                                      bField_,
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      circle_fit_resultsGPU_,
                                                                      offset);
    cudaCheck(hipGetLastError());

    kernelLineFit<3><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                    3,
                                                                    bField_,
                                                                    outputSoa_d,
                                                                    hitsGPU_.get(),
                                                                    hits_geGPU_.get(),
                                                                    fast_fit_resultsGPU_.get(),
                                                                    circle_fit_resultsGPU_,
                                                                    offset);
    cudaCheck(hipGetLastError());

    // quads
    kernelFastFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tuples_d,
                                                                    tupleMultiplicity_d,
                                                                    4,
                                                                    hh.view(),
                                                                    hitsGPU_.get(),
                                                                    hits_geGPU_.get(),
                                                                    fast_fit_resultsGPU_.get(),
                                                                    offset);
    cudaCheck(hipGetLastError());

    kernelCircleFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                      4,
                                                                      bField_,
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      circle_fit_resultsGPU_,
                                                                      offset);
    cudaCheck(hipGetLastError());

    kernelLineFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                    4,
                                                                    bField_,
                                                                    outputSoa_d,
                                                                    hitsGPU_.get(),
                                                                    hits_geGPU_.get(),
                                                                    fast_fit_resultsGPU_.get(),
                                                                    circle_fit_resultsGPU_,
                                                                    offset);
    cudaCheck(hipGetLastError());

    if (fit5as4_) {
      // penta
      kernelFastFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tuples_d,
                                                                      tupleMultiplicity_d,
                                                                      5,
                                                                      hh.view(),
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      offset);
      cudaCheck(hipGetLastError());

      kernelCircleFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                        5,
                                                                        bField_,
                                                                        hitsGPU_.get(),
                                                                        hits_geGPU_.get(),
                                                                        fast_fit_resultsGPU_.get(),
                                                                        circle_fit_resultsGPU_,
                                                                        offset);
      cudaCheck(hipGetLastError());

      kernelLineFit<4><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                      5,
                                                                      bField_,
                                                                      outputSoa_d,
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      circle_fit_resultsGPU_,
                                                                      offset);
      cudaCheck(hipGetLastError());
    } else {
      // penta all 5
      kernelFastFit<5><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tuples_d,
                                                                      tupleMultiplicity_d,
                                                                      5,
                                                                      hh.view(),
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      offset);
      cudaCheck(hipGetLastError());

      kernelCircleFit<5><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                        5,
                                                                        bField_,
                                                                        hitsGPU_.get(),
                                                                        hits_geGPU_.get(),
                                                                        fast_fit_resultsGPU_.get(),
                                                                        circle_fit_resultsGPU_,
                                                                        offset);
      cudaCheck(hipGetLastError());

      kernelLineFit<5><<<numberOfBlocks, blockSize, 0, stream.id()>>>(tupleMultiplicity_d,
                                                                      5,
                                                                      bField_,
                                                                      outputSoa_d,
                                                                      hitsGPU_.get(),
                                                                      hits_geGPU_.get(),
                                                                      fast_fit_resultsGPU_.get(),
                                                                      circle_fit_resultsGPU_,
                                                                      offset);
      cudaCheck(hipGetLastError());
    }
  }
}
