#include "hip/hip_runtime.h"
#include "test_common.h"
#include <iostream>

#include <Eigen/Core>
#include <Eigen/Eigenvalues>


using namespace Eigen;

__host__ __device__ void eigenValues(Matrix3d * m, Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType * ret) {
  if (!NODEBUG) {
    printf("Matrix(0,0): %f\n", (*m)(0,0));
    printf("Matrix(1,1): %f\n", (*m)(1,1));
    printf("Matrix(2,2): %f\n", (*m)(2,2));
  }
  SelfAdjointEigenSolver<Matrix3d> es;
  es.computeDirect(*m);
  (*ret) = es.eigenvalues();
  return;
}

__global__ void kernel(Matrix3d * m, Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType * ret) {
  eigenValues(m, ret);
}

__global__ void kernelInverse(Matrix3d * in, Matrix3d * out) {
//  (*out) = in->inverse();
}

template<typename M1, typename M2, typename M3>
__global__ void kernelMultiply(M1 * J,
                               M2 * C,
                               M3 * result) {
//  Map<M3> res(result->data());
  if (!NODEBUG)
    printf("*** GPU IN ***\n");
  printIt(J);
  printIt(C);
//  res.noalias() = (*J) * (*C);
//  printIt(&res);
  (*result) = (*J) * (*C);
  if (!NODEBUG)
    printf("*** GPU OUT ***\n");
  return;
}

template<int row1, int col1, int row2, int col2>
void testMultiply() {
  std::cout << "TEST MULTIPLY" << std::endl;
  std::cout << "Product of type " << row1 << "x" << col1
    << " * " << row2 << "x" << col2 << std::endl;
  Eigen::Matrix<double, row1, col1> J;
  fillMatrix(J);
  Eigen::Matrix<double, row2, col2> C;
  fillMatrix(C);
  Eigen::Matrix<double, row1, col2> multiply_result = J * C;
  if (!NODEBUG) {
    std::cout << "Input J:" << std::endl; printIt(&J);
    std::cout << "Input C:" << std::endl; printIt(&C);
    std::cout << "Output:" << std::endl;
    printIt(&multiply_result);
  }
  // GPU
  Eigen::Matrix<double, row1, col1> *JGPU = nullptr;
  Eigen::Matrix<double, row2, col2> *CGPU = nullptr;
  Eigen::Matrix<double, row1, col2> *multiply_resultGPU = nullptr;
  Eigen::Matrix<double, row1, col2> *multiply_resultGPUret = new Eigen::Matrix<double, row1, col2>();

  hipMalloc((void **)&JGPU, sizeof(Eigen::Matrix<double, row1, col1>));
  hipMalloc((void **)&CGPU, sizeof(Eigen::Matrix<double, row2, col2>));
  hipMalloc((void **)&multiply_resultGPU, sizeof(Eigen::Matrix<double, row1, col2>));
  hipMemcpy(JGPU, &J, sizeof(Eigen::Matrix<double, row1, col1>), hipMemcpyHostToDevice);
  hipMemcpy(CGPU, &C, sizeof(Eigen::Matrix<double, row2, col2>), hipMemcpyHostToDevice);
  hipMemcpy(multiply_resultGPU, &multiply_result, sizeof(Eigen::Matrix<double, row1, col2>), hipMemcpyHostToDevice);

  kernelMultiply<<<1,1>>>(JGPU, CGPU, multiply_resultGPU);
  hipDeviceSynchronize();

  hipMemcpy(multiply_resultGPUret, multiply_resultGPU, 
      sizeof(Eigen::Matrix<double, row1, col2>), hipMemcpyDeviceToHost);
  printIt(multiply_resultGPUret);
  assert(isEqualFuzzy(multiply_result, (*multiply_resultGPUret)));
}

void testInverse() {
  std::cout << "TEST INVERSE" << std::endl;
  Matrix3d m = Matrix3d::Random();
  Matrix3d *mGPU = nullptr;
  Matrix3d *mGPUret = nullptr;
  Matrix3d *mCPUret = new Matrix3d();

  if (!NODEBUG) {
    std::cout << "Here is the matrix m:" << std::endl << m << std::endl;
    std::cout << "Its inverse is:" << std::endl << m.inverse() << std::endl;
  }
  hipMalloc((void **)&mGPU, sizeof(Matrix3d));
  hipMalloc((void **)&mGPUret, sizeof(Matrix3d));
  hipMemcpy(mGPU, &m, sizeof(Matrix3d), hipMemcpyHostToDevice);

  kernelInverse<<<1,1>>>(mGPU, mGPUret);
  hipDeviceSynchronize();

  hipMemcpy(mCPUret, mGPUret, sizeof(Matrix3d), hipMemcpyDeviceToHost);
  if (!NODEBUG)
    std::cout << "Its GPU inverse is:" << std::endl << (*mCPUret) << std::endl;
}

void testEigenvalues() {
  std::cout << "TEST EIGENVALUES" << std::endl;
  Matrix3d m = Matrix3d::Random();
  Matrix3d mt = m.transpose();
  m += mt;
  Matrix3d * m_gpu = nullptr;
  Matrix3d * mgpudebug = new Matrix3d();
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret = new Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType;
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret1 = new Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType;
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret_gpu = nullptr;
  eigenValues(&m, ret);
  if (!NODEBUG) {
    std::cout << "Generated Matrix M 3x3:\n" << m << std::endl;
    std::cout << "The eigenvalues of M are:" << std::endl << (*ret) << std::endl;
    std::cout << "*************************\n\n" << std::endl;
  }
  hipMalloc((void **)&m_gpu, sizeof(Matrix3d));
  hipMalloc((void **)&ret_gpu, sizeof(Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType));
  hipMemcpy(m_gpu, &m, sizeof(Matrix3d), hipMemcpyHostToDevice);

  kernel<<<1,1>>>(m_gpu, ret_gpu);
  hipDeviceSynchronize();

  hipMemcpy(mgpudebug, m_gpu, sizeof(Matrix3d), hipMemcpyDeviceToHost);
  hipMemcpy(ret1, ret_gpu, sizeof(Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType), hipMemcpyDeviceToHost);
  if (!NODEBUG) {
    std::cout << "GPU Generated Matrix M 3x3:\n" << (*mgpudebug) << std::endl;
    std::cout << "GPU The eigenvalues of M are:" << std::endl << (*ret1) << std::endl;
    std::cout << "*************************\n\n" << std::endl;
  }
  assert(isEqualFuzzy(*ret, *ret1));
}


int main (int argc, char * argv[]) {

  testEigenvalues();
  testInverse();
  testMultiply<1, 2, 2, 1>();
  testMultiply<1, 2, 2, 2>();
  testMultiply<1, 2, 2, 3>();
  testMultiply<1, 2, 2, 4>();
  testMultiply<1, 2, 2, 5>();
  testMultiply<2, 1, 1, 2>();
  testMultiply<2, 1, 1, 3>();
  testMultiply<2, 1, 1, 4>();
  testMultiply<2, 1, 1, 5>();
  testMultiply<2, 2, 2, 2>();
  testMultiply<2, 3, 3, 1>();
  testMultiply<2, 3, 3, 2>();
  testMultiply<2, 3, 3, 4>();
  testMultiply<2, 3, 3, 5>();
  testMultiply<3, 2, 2, 3>();
  testMultiply<2, 3, 3, 3>(); // DOES NOT COMPILE W/O PATCHING EIGEN
  testMultiply<3, 3, 3, 3>();
  testMultiply<8, 8, 8, 8>();
  testMultiply<3, 4, 4, 3>();
  testMultiply<2, 4, 4, 2>();
  testMultiply<3, 4, 4, 2>(); // DOES NOT COMPILE W/O PATCHING EIGEN

  return 0;
}
