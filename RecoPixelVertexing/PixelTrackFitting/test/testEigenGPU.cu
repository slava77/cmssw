#include "hip/hip_runtime.h"
#include <iostream>

#include <Eigen/Core>
#include <Eigen/Eigenvalues>

#include "RecoPixelVertexing/PixelTrackFitting/interface/RiemannFit.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"

#include "test_common.h"

using namespace Eigen;

__global__
void kernelFullFit(Rfit::Matrix3xNd * hits,
    Rfit::Matrix3Nd * hits_cov,
    double B,
    bool errors,
    Rfit::circle_fit * circle_fit_resultsGPU,
    Rfit::line_fit * line_fit_resultsGPU) {

  printf("hits size: %d,%d\n", hits->rows(), hits->cols());
  Rfit::printIt(hits, "KernelFulFit - input hits: ");
  Vector4d fast_fit = Rfit::Fast_fit(*hits);

  u_int n = hits->cols();
  Rfit::VectorNd rad = (hits->block(0, 0, 2, n).colwise().norm());

  Rfit::Matrix2xNd hits2D_local = (hits->block(0,0,2,n)).eval();
  Rfit::Matrix2Nd hits_cov2D_local = (hits_cov->block(0, 0, 2 * n, 2 * n)).eval();
  Rfit::printIt(&hits2D_local, "kernelFullFit - hits2D_local: ");
  Rfit::printIt(&hits_cov2D_local, "kernelFullFit - hits_cov2D_local: ");
  printf("kernelFullFit - hits address: %p\n", hits);
  printf("kernelFullFit - hits_cov address: %p\n", hits_cov);
  printf("kernelFullFit - hits_cov2D address: %p\n", &hits2D_local);
  printf("kernelFullFit - hits_cov2D_local address: %p\n", &hits_cov2D_local);
  /* At some point I gave up and locally construct block on the stack, so that
     the next invocation to Rfit::Circle_fit works properly. Failing to do so
     implied basically an empty collection of hits and covariances. That could
     have been partially fixed if values of the passed in matrices would have
     been printed on screen since that, maybe, triggered internally the real
     creations of the blocks. To be understood and compared against the myriad
     of compilation warnings we have.
     */

  (*circle_fit_resultsGPU) =
    Rfit::Circle_fit(hits->block(0,0,2,n), hits_cov->block(0, 0, 2 * n, 2 * n),
        fast_fit, rad, B, errors);
  /*
     (*circle_fit_resultsGPU) =
     Rfit::Circle_fit(hits2D_local, hits_cov2D_local,
     fast_fit, rad, B, errors);
  */
  (*line_fit_resultsGPU) = Rfit::Line_fit(*hits, *hits_cov, *circle_fit_resultsGPU, fast_fit, B, errors);

  return;
}

void fillHitsAndHitsCov(Rfit::Matrix3xNd & hits, Rfit::Matrix3Nd & hits_cov) {
  hits << 1.98645, 4.72598, 7.65632, 11.3151,
          2.18002, 4.88864, 7.75845, 11.3134,
          2.46338, 6.99838,  11.808,  17.793;
  hits_cov(0,0) = 7.14652e-06;
  hits_cov(1,1) = 2.15789e-06;
  hits_cov(2,2) = 1.63328e-06;
  hits_cov(3,3) = 6.27919e-06;
  hits_cov(4,4) = 6.10348e-06;
  hits_cov(5,5) = 2.08211e-06;
  hits_cov(6,6) = 1.61672e-06;
  hits_cov(7,7) = 6.28081e-06;
  hits_cov(8,8) = 5.184e-05;
  hits_cov(9,9) = 1.444e-05;
  hits_cov(10,10) = 6.25e-06;
  hits_cov(11,11) = 3.136e-05;
  hits_cov(0,4) = hits_cov(4,0) = -5.60077e-06;
  hits_cov(1,5) = hits_cov(5,1) = -1.11936e-06;
  hits_cov(2,6) = hits_cov(6,2) = -6.24945e-07;
  hits_cov(3,7) = hits_cov(7,3) = -5.28e-06;
}

void testFitOneGo(bool errors, double epsilon=1e-6) {
  constexpr double B = 0.0113921;
  Rfit::Matrix3xNd hits(3,4);
  Rfit::Matrix3Nd hits_cov = MatrixXd::Zero(12,12);

  fillHitsAndHitsCov(hits, hits_cov);

  // FAST_FIT_CPU
  Vector4d fast_fit_results = Rfit::Fast_fit(hits);
  // CIRCLE_FIT CPU
  u_int n = hits.cols();
  Rfit::VectorNd rad = (hits.block(0, 0, 2, n).colwise().norm());

  Rfit::circle_fit circle_fit_results = Rfit::Circle_fit(hits.block(0, 0, 2, n),
      hits_cov.block(0, 0, 2 * n, 2 * n),
      fast_fit_results, rad, B, errors);
  // LINE_FIT CPU
  Rfit::line_fit line_fit_results = Rfit::Line_fit(hits, hits_cov, circle_fit_results,
      fast_fit_results, B, errors);

  // FIT GPU
  std::cout << "GPU FIT" << std::endl;
  Rfit::Matrix3xNd * hitsGPU = nullptr; // new Rfit::Matrix3xNd(3,4);
  Rfit::Matrix3Nd * hits_covGPU = nullptr;
  Rfit::line_fit * line_fit_resultsGPU = nullptr;
  Rfit::line_fit * line_fit_resultsGPUret = new Rfit::line_fit();
  Rfit::circle_fit * circle_fit_resultsGPU = nullptr; // new Rfit::circle_fit();
  Rfit::circle_fit * circle_fit_resultsGPUret = new Rfit::circle_fit();

  cudaCheck(hipMalloc((void **)&hitsGPU, sizeof(Rfit::Matrix3xNd(3,4))));
  cudaCheck(hipMalloc((void **)&hits_covGPU, sizeof(Rfit::Matrix3Nd(12,12))));
  cudaCheck(hipMalloc((void **)&line_fit_resultsGPU, sizeof(Rfit::line_fit)));
  cudaCheck(hipMalloc((void **)&circle_fit_resultsGPU, sizeof(Rfit::circle_fit)));
  cudaCheck(hipMemcpy(hitsGPU, &hits, sizeof(Rfit::Matrix3xNd(3,4)), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(hits_covGPU, &hits_cov, sizeof(Rfit::Matrix3Nd(12,12)), hipMemcpyHostToDevice));

  kernelFullFit<<<1, 1>>>(hitsGPU, hits_covGPU, B, errors,
      circle_fit_resultsGPU, line_fit_resultsGPU);
  cudaCheck(hipDeviceSynchronize());

  cudaCheck(hipMemcpy(circle_fit_resultsGPUret, circle_fit_resultsGPU, sizeof(Rfit::circle_fit), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(line_fit_resultsGPUret, line_fit_resultsGPU, sizeof(Rfit::line_fit), hipMemcpyDeviceToHost));

  std::cout << "Fitted values (CircleFit) CPU:\n" << circle_fit_results.par << std::endl;
  std::cout << "Fitted values (LineFit): CPU\n" << line_fit_results.par << std::endl;
  std::cout << "Fitted values (CircleFit) GPU:\n" << circle_fit_resultsGPUret->par << std::endl;
  std::cout << "Fitted values (LineFit): GPU\n" << line_fit_resultsGPUret->par << std::endl;
  assert(isEqualFuzzy(circle_fit_results.par, circle_fit_resultsGPUret->par, epsilon));
  assert(isEqualFuzzy(line_fit_results.par, line_fit_resultsGPUret->par, epsilon));

  cudaCheck(hipFree(hitsGPU));
  cudaCheck(hipFree(hits_covGPU));
  cudaCheck(hipFree(line_fit_resultsGPU));
  cudaCheck(hipFree(circle_fit_resultsGPU));
  delete line_fit_resultsGPUret;
  delete circle_fit_resultsGPUret;

  hipDeviceReset();
}

int main (int argc, char * argv[]) {

  hipDeviceSetLimit(hipLimitStackSize, 32*1024);
  std::cout << "TEST FIT, ERRORS AND SCATTER" << std::endl;
  testFitOneGo(true, 1e-5);

  return 0;
}

