#include "hip/hip_runtime.h"
// C++ headers
#include <algorithm>
#include <numeric>

// CUDA runtime
#include <hip/hip_runtime.h>

// CMSSW headers
#include "RecoLocalTracker/SiPixelClusterizer/plugins/SiPixelRawToClusterGPUKernel.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuClustering.h"
#include "PixelRecHits.h"
#include "gpuPixelRecHits.h"

namespace {
  __global__
  void setHitsLayerStart(const uint32_t *hitsModuleStart, const uint32_t *layerStart, uint32_t *hitsLayerStart) {
    auto i = blockIdx.x*blockDim.x + threadIdx.x;

    if(i < 10) {
      hitsLayerStart[i] = hitsModuleStart[layerStart[i]];
    }
    else if(i == 10) {
      hitsLayerStart[i] = hitsModuleStart[gpuClustering::MaxNumModules];
    }
  }
}

namespace pixelgpudetails {
  PixelRecHitGPUKernel::PixelRecHitGPUKernel(cuda::stream_t<>& cudaStream) {

    cudaCheck(hipMalloc((void**) & gpu_.bs_d,3*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.hitsLayerStart_d,(11)*sizeof(uint32_t)));
    cudaCheck(hipMalloc((void**) & gpu_.charge_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.detInd_d,(gpuClustering::MaxNumModules*256)*sizeof(uint16_t)));
    cudaCheck(hipMalloc((void**) & gpu_.xg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.yg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.zg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.rg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.xl_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.yl_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.xerr_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.yerr_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.iphi_d,(gpuClustering::MaxNumModules*256)*sizeof(int16_t)));
    cudaCheck(hipMalloc((void**) & gpu_.sortIndex_d,(gpuClustering::MaxNumModules*256)*sizeof(uint16_t)));
    cudaCheck(hipMalloc((void**) & gpu_.mr_d,(gpuClustering::MaxNumModules*256)*sizeof(uint16_t)));
    cudaCheck(hipMalloc((void**) & gpu_.mc_d,(gpuClustering::MaxNumModules*256)*sizeof(uint16_t)));
    cudaCheck(hipMalloc((void**) & gpu_.hist_d, 10*sizeof(HitsOnGPU::Hist)));
    cudaCheck(hipMalloc((void**) & gpu_d, sizeof(HitsOnGPU)));
    gpu_.me_d = gpu_d;
    cudaCheck(hipMemcpyAsync(gpu_d, &gpu_, sizeof(HitsOnGPU), hipMemcpyDefault,cudaStream.id()));

    // Feels a bit dumb but constexpr arrays are not supported for device code
    // TODO: should be moved to EventSetup (or better ideas?)
    // Would it be better to use "constant memory"?
    cudaCheck(hipMalloc((void**) & d_phase1TopologyLayerStart_, 11*sizeof(uint32_t)));
    cudaCheck(hipMemcpyAsync(d_phase1TopologyLayerStart_, phase1PixelTopology::layerStart, 11*sizeof(uint32_t), hipMemcpyDefault, cudaStream.id()));
  }

  PixelRecHitGPUKernel::~PixelRecHitGPUKernel() {
    cudaCheck(hipFree(gpu_.charge_d));
    cudaCheck(hipFree(gpu_.detInd_d));
    cudaCheck(hipFree(gpu_.xg_d));
    cudaCheck(hipFree(gpu_.yg_d));
    cudaCheck(hipFree(gpu_.zg_d));
    cudaCheck(hipFree(gpu_.rg_d));
    cudaCheck(hipFree(gpu_.xl_d));
    cudaCheck(hipFree(gpu_.yl_d));
    cudaCheck(hipFree(gpu_.xerr_d));
    cudaCheck(hipFree(gpu_.yerr_d));
    cudaCheck(hipFree(gpu_.iphi_d));
    cudaCheck(hipFree(gpu_.sortIndex_d));
    cudaCheck(hipFree(gpu_.mr_d));
    cudaCheck(hipFree(gpu_.mc_d));
    cudaCheck(hipFree(gpu_.hist_d));
    cudaCheck(hipFree(gpu_d));

    cudaCheck(hipFree(d_phase1TopologyLayerStart_));
  }

  void PixelRecHitGPUKernel::makeHitsAsync(const siPixelRawToClusterHeterogeneousProduct::GPUProduct& input,
                                           float const * bs,
                                           pixelCPEforGPU::ParamsOnGPU const * cpeParams,
                                           cuda::stream_t<>& stream) {
   cudaCheck(hipMemcpyAsync(gpu_.bs_d, bs, 3*sizeof(float), hipMemcpyDefault, stream.id()));
   gpu_.hitsModuleStart_d = input.clusModuleStart_d;
   cudaCheck(hipMemcpyAsync(gpu_d, &gpu_, sizeof(HitsOnGPU), hipMemcpyDefault, stream.id()));

    int threadsPerBlock = 256;
    int blocks = input.nModules; // active modules (with digis)
    gpuPixelRecHits::getHits<<<blocks, threadsPerBlock, 0, stream.id()>>>(
      cpeParams,
      gpu_.bs_d,
      input.moduleInd_d,
      input.xx_d, input.yy_d, input.adc_d,
      input.moduleStart_d,
      input.clusInModule_d, input.moduleId_d,
      input.clus_d,
      input.nDigis,
      gpu_.hitsModuleStart_d,
      gpu_.charge_d,
      gpu_.detInd_d,
      gpu_.xg_d, gpu_.yg_d, gpu_.zg_d, gpu_.rg_d,
      gpu_.iphi_d,
      gpu_.xl_d, gpu_.yl_d,
      gpu_.xerr_d, gpu_.yerr_d,
      gpu_.mr_d, gpu_.mc_d
    );

    // assuming full warp of threads is better than a smaller number...
    setHitsLayerStart<<<1, 32, 0, stream.id()>>>(gpu_.hitsModuleStart_d, d_phase1TopologyLayerStart_, gpu_.hitsLayerStart_d);

    // needed only if hits on CPU are required...
    cudaCheck(hipMemcpyAsync(hitsModuleStart_, gpu_.hitsModuleStart_d, (gpuClustering::MaxNumModules+1) * sizeof(uint32_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(hitsLayerStart_, gpu_.hitsLayerStart_d, 11*sizeof(uint32_t), hipMemcpyDefault, stream.id()));
    auto nhits = input.nClusters;
    cpu_ = std::make_unique<HitsOnCPU>(nhits);
    cudaCheck(hipMemcpyAsync(cpu_->charge.data(), gpu_.charge_d, nhits*sizeof(int32_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(cpu_->xl.data(), gpu_.xl_d, nhits*sizeof(float), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(cpu_->yl.data(), gpu_.yl_d, nhits*sizeof(float), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(cpu_->xe.data(), gpu_.xerr_d, nhits*sizeof(float), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(cpu_->ye.data(), gpu_.yerr_d, nhits*sizeof(float), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(cpu_->mr.data(), gpu_.mr_d, nhits*sizeof(uint16_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(cpu_->mc.data(), gpu_.mc_d, nhits*sizeof(uint16_t), hipMemcpyDefault, stream.id()));

#ifdef GPU_DEBUG
    hipStreamSynchronize(stream.id());

    std::cout << "hit layerStart ";
    for (int i=0;i<10;++i) std::cout << phase1PixelTopology::layerName[i] << ':' << hitsLayerStart_[i] << ' ';
    std::cout << "end:" << hitsLayerStart_[10] << std::endl;
#endif

    // for timing test
    // hipStreamSynchronize(stream.id());
    // auto nhits = hitsLayerStart_[10];
    // radixSortMultiWrapper<int16_t><<<10, 256, 0, c.stream>>>(gpu_.iphi_d,gpu_.sortIndex_d,gpu_.hitsLayerStart_d);

    cudautils::fillManyFromVector(gpu_.hist_d,10,gpu_.iphi_d, gpu_.hitsLayerStart_d, nhits,256,stream.id());
  }

  std::unique_ptr<HitsOnCPU>&& PixelRecHitGPUKernel::getOutput(cuda::stream_t<>& stream) {
    cpu_->gpu_d = gpu_d;
    memcpy(cpu_->hitsModuleStart, hitsModuleStart_, (gpuClustering::MaxNumModules+1) * sizeof(uint32_t));
    return std::move(cpu_);
  }
}
