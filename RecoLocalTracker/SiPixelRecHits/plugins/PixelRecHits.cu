// C++ headers
#include <algorithm>
#include <numeric>

// CUDA runtime
#include <hip/hip_runtime.h>

// thrust heders
#include <thrust/scan.h>
#include <thrust/system/cuda/execution_policy.h>

// CMSSW headers
#include "RecoLocalTracker/SiPixelClusterizer/plugins/SiPixelRawToClusterGPUKernel.h"
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuClustering.h"
#include "PixelRecHits.h"
#include "gpuPixelRecHits.h"

namespace pixelgpudetails {
  PixelRecHitGPUKernel::PixelRecHitGPUKernel(cuda::stream_t<>& cudaStream) {

    cudaCheck(hipMalloc((void**) & gpu_.bs_d,3*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.hitsModuleStart_d,(gpuClustering::MaxNumModules+1)*sizeof(uint32_t)));
    cudaCheck(hipMalloc((void**) & gpu_.hitsLayerStart_d,(11)*sizeof(uint32_t)));
    cudaCheck(hipMalloc((void**) & gpu_.charge_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.detInd_d,(gpuClustering::MaxNumModules*256)*sizeof(uint16_t)));
    cudaCheck(hipMalloc((void**) & gpu_.xg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.yg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.zg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.rg_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.xl_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.yl_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.xerr_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.yerr_d,(gpuClustering::MaxNumModules*256)*sizeof(float)));
    cudaCheck(hipMalloc((void**) & gpu_.iphi_d,(gpuClustering::MaxNumModules*256)*sizeof(int16_t)));
    cudaCheck(hipMalloc((void**) & gpu_.sortIndex_d,(gpuClustering::MaxNumModules*256)*sizeof(uint16_t)));
    cudaCheck(hipMalloc((void**) & gpu_.mr_d,(gpuClustering::MaxNumModules*256)*sizeof(uint16_t)));
    cudaCheck(hipMalloc((void**) & gpu_.mc_d,(gpuClustering::MaxNumModules*256)*sizeof(uint16_t)));
//    cudaCheck(hipMalloc((void**) & gpu_.hist_d, 10*sizeof(HitsOnGPU::Hist)));

    cudaCheck(hipMalloc((void**) & gpu_d, sizeof(HitsOnGPU)));
    cudaCheck(hipMemcpyAsync(gpu_d, &gpu_, sizeof(HitsOnGPU), hipMemcpyDefault,cudaStream.id()));

  }

  PixelRecHitGPUKernel::~PixelRecHitGPUKernel() {
    cudaCheck(hipFree(gpu_.hitsModuleStart_d));
    cudaCheck(hipFree(gpu_.charge_d));
    cudaCheck(hipFree(gpu_.detInd_d));
    cudaCheck(hipFree(gpu_.xg_d));
    cudaCheck(hipFree(gpu_.yg_d));
    cudaCheck(hipFree(gpu_.zg_d));
    cudaCheck(hipFree(gpu_.rg_d));
    cudaCheck(hipFree(gpu_.xl_d));
    cudaCheck(hipFree(gpu_.yl_d));
    cudaCheck(hipFree(gpu_.xerr_d));
    cudaCheck(hipFree(gpu_.yerr_d));
    cudaCheck(hipFree(gpu_.iphi_d));
    cudaCheck(hipFree(gpu_.sortIndex_d));
    cudaCheck(hipFree(gpu_.mr_d));
    cudaCheck(hipFree(gpu_.mc_d));
    // cudaCheck(hipFree(gpu_.hist_d));

    cudaCheck(hipFree(gpu_d));
  }

  void PixelRecHitGPUKernel::makeHitsAsync(const siPixelRawToClusterHeterogeneousProduct::GPUProduct& input,
                                           float const * bs,
                                           pixelCPEforGPU::ParamsOnGPU const * cpeParams,
                                           cuda::stream_t<>& stream) {

   cudaCheck(hipMemcpyAsync(gpu_.bs_d, bs, 3*sizeof(float), hipMemcpyDefault, stream.id()));

    thrust::exclusive_scan(thrust::cuda::par.on(stream.id()),
                           input.clusInModule_d,
                           input.clusInModule_d + gpuClustering::MaxNumModules + 1,
                           gpu_.hitsModuleStart_d);
  
    int threadsPerBlock = 256;
    int blocks = input.nModules; // active modules (with digis)
    gpuPixelRecHits::getHits<<<blocks, threadsPerBlock, 0, stream.id()>>>(
      cpeParams,
      gpu_.bs_d,
      input.moduleInd_d,
      input.xx_d, input.yy_d, input.adc_d,
      input.moduleStart_d,
      input.clusInModule_d, input.moduleId_d,
      input.clus_d,
      input.nDigis,
      gpu_.hitsModuleStart_d,
      gpu_.charge_d,
      gpu_.detInd_d,
      gpu_.xg_d, gpu_.yg_d, gpu_.zg_d, gpu_.rg_d,
      gpu_.iphi_d,
      gpu_.xl_d, gpu_.yl_d,
      gpu_.xerr_d, gpu_.yerr_d, 
      gpu_.mr_d, gpu_.mc_d
    );

    // needed only if hits on CPU are required...
    cudaCheck(hipMemcpyAsync(hitsModuleStart_, gpu_.hitsModuleStart_d, (gpuClustering::MaxNumModules+1) * sizeof(uint32_t), hipMemcpyDefault, stream.id()));
   
    // to be moved to gpu?
    auto nhits = hitsModuleStart_[gpuClustering::MaxNumModules];
    for (int i=0;i<10;++i) hitsLayerStart_[i]=hitsModuleStart_[phase1PixelTopology::layerStart[i]];
    hitsLayerStart_[10]=nhits;

    std::cout << "hit layerStart "; 
    for (int i=0;i<10;++i) std::cout << phase1PixelTopology::layerName[i] << ':' << hitsLayerStart_[i] << ' ';
    std::cout << "end:" << hitsLayerStart_[10] << std::endl;

    cudaCheck(hipMemcpyAsync(gpu_.hitsLayerStart_d, hitsLayerStart_, (11) * sizeof(uint32_t), hipMemcpyDefault, stream.id()));

    // for timing test
    // radixSortMultiWrapper<int16_t><<<10, 256, 0, c.stream>>>(gpu_.iphi_d,gpu_.sortIndex_d,gpu_.hitsLayerStart_d);

    // fillManyFromVector(gpu_.hist_d,10,gpu_.iphi_d, gpu_.hitsLayerStart_d, nhits,256,c.stream);


  }

  HitsOnCPU PixelRecHitGPUKernel::getOutput(cuda::stream_t<>& stream) const {
    // needed only if hits on CPU are required...
    auto nhits = hitsModuleStart_[gpuClustering::MaxNumModules];

    HitsOnCPU hoc(nhits);
    hoc.gpu_d = gpu_d;
    memcpy(hoc.hitsModuleStart, hitsModuleStart_, (gpuClustering::MaxNumModules+1) * sizeof(uint32_t));
    cudaCheck(hipMemcpyAsync(hoc.charge.data(), gpu_.charge_d, nhits*sizeof(uint32_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(hoc.xl.data(), gpu_.xl_d, nhits*sizeof(uint32_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(hoc.yl.data(), gpu_.yl_d, nhits*sizeof(uint32_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(hoc.xe.data(), gpu_.xerr_d, nhits*sizeof(uint32_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(hoc.ye.data(), gpu_.yerr_d, nhits*sizeof(uint32_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(hoc.mr.data(), gpu_.mr_d, nhits*sizeof(uint16_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(hoc.mc.data(), gpu_.mc_d, nhits*sizeof(uint16_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipStreamSynchronize(stream.id()));
    return hoc;
  }
}
