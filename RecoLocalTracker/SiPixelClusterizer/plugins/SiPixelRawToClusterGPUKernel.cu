#include "hip/hip_runtime.h"
/* Sushil Dubey, Shashi Dugad, TIFR, July 2017
 *
 * File Name: RawToClusterGPU.cu
 * Description: It converts Raw data into Digi Format on GPU
 * then it converts adc -> electron and
 * applies the adc threshold to needed for clustering
 * Finaly the Output of RawToDigi data is given to pixelClusterizer
 *
**/

// C++ includes
#include <cassert>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <string>

// CUDA includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>

// cub includes
#include <hipcub/hipcub.hpp>

// CMSSW includes
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuCalibPixel.h"
#include "RecoLocalTracker/SiPixelClusterizer/plugins/gpuClustering.h"
#include "RecoLocalTracker/SiPixelClusterizer/interface/SiPixelFedCablingMapGPU.h"

// local includes
#include "SiPixelRawToClusterGPUKernel.h"

namespace pixelgpudetails {

  SiPixelRawToClusterGPUKernel::SiPixelRawToClusterGPUKernel(cuda::stream_t<>& cudaStream) {
    int WSIZE = pixelgpudetails::MAX_FED * pixelgpudetails::MAX_WORD;
    hipHostMalloc(&word,       sizeof(unsigned int)*WSIZE);
    hipHostMalloc(&fedId_h,    sizeof(unsigned char)*WSIZE);

    // to store the output of RawToDigi
    hipHostMalloc(&pdigi_h,    sizeof(uint32_t)*WSIZE);
    hipHostMalloc(&rawIdArr_h, sizeof(uint32_t)*WSIZE);

    hipHostMalloc(&adc_h, sizeof(uint16_t)*WSIZE);
    hipHostMalloc(&clus_h, sizeof(int32_t)*WSIZE);

    constexpr uint32_t vsize = sizeof(GPU::SimpleVector<pixelgpudetails::error_obj>);
    constexpr uint32_t esize = sizeof(pixelgpudetails::error_obj);
    cudaCheck(hipHostMalloc(&error_h, vsize));
    cudaCheck(hipHostMalloc(&error_h_tmp, vsize));
    cudaCheck(hipHostMalloc(&data_h, MAX_FED*pixelgpudetails::MAX_WORD*esize));

    new (error_h) GPU::SimpleVector<pixelgpudetails::error_obj>(MAX_FED*pixelgpudetails::MAX_WORD, data_h);
    new (error_h_tmp) GPU::SimpleVector<pixelgpudetails::error_obj>(MAX_FED*pixelgpudetails::MAX_WORD, data_d);
    assert(error_h->size() == 0);
    assert(error_h->capacity() == static_cast<int>(MAX_FED*pixelgpudetails::MAX_WORD));
    assert(error_h_tmp->size() == 0);
    assert(error_h_tmp->capacity() == static_cast<int>(MAX_FED*pixelgpudetails::MAX_WORD));

    // allocate memory for RawToDigi on GPU
    using namespace gpuClustering;

    // Number of words for all the feds
    constexpr uint32_t MAX_WORD08_SIZE = MAX_FED * pixelgpudetails::MAX_WORD  * sizeof(uint8_t);
    constexpr uint32_t MAX_WORD32_SIZE = MAX_FED * pixelgpudetails::MAX_WORD  * sizeof(uint32_t);
    constexpr uint32_t MAX_WORD16_SIZE = MAX_FED * pixelgpudetails::MAX_WORD  * sizeof(uint16_t);
    constexpr uint32_t MAX_ERROR_SIZE  = MAX_FED * pixelgpudetails::MAX_WORD * esize;

    cudaCheck(hipMalloc((void**) & word_d,        MAX_WORD32_SIZE));
    cudaCheck(hipMalloc((void**) & fedId_d,       MAX_WORD08_SIZE));
    cudaCheck(hipMalloc((void**) & pdigi_d,       MAX_WORD32_SIZE)); // to store thepacked digi
    cudaCheck(hipMalloc((void**) & xx_d,          MAX_WORD16_SIZE)); // to store the x and y coordinate
    cudaCheck(hipMalloc((void**) & yy_d,          MAX_WORD16_SIZE));
    cudaCheck(hipMalloc((void**) & adc_d,         MAX_WORD16_SIZE));

    cudaCheck(hipMalloc((void**) & moduleInd_d,   MAX_WORD16_SIZE));
    cudaCheck(hipMalloc((void**) & rawIdArr_d,    MAX_WORD32_SIZE));
    cudaCheck(hipMalloc((void**) & error_d,       vsize));
    cudaCheck(hipMalloc((void**) & data_d,        MAX_ERROR_SIZE));

    // for the clusterizer
    cudaCheck(hipMalloc((void**) & clus_d,        MAX_WORD32_SIZE)); // cluser index in module

    cudaCheck(hipMalloc((void**) & moduleStart_d, (MaxNumModules+1)*sizeof(uint32_t) ));
    cudaCheck(hipMalloc((void**) & clusInModule_d,(MaxNumModules)*sizeof(uint32_t) ));
    cudaCheck(hipMalloc((void**) & moduleId_d,    (MaxNumModules)*sizeof(uint32_t) ));

    cudaCheck(hipMalloc((void**) & gpuProduct_d, sizeof(GPUProduct)));
    gpuProduct = getProduct();
    assert(xx_d==gpuProduct.xx_d);

    cudaCheck(hipMemcpyAsync(gpuProduct_d, &gpuProduct, sizeof(GPUProduct), hipMemcpyDefault,cudaStream.id()));

    // originally from rechits
    cudaCheck(hipMalloc((void**) & clusModuleStart_d, (MaxNumModules+1)*sizeof(uint32_t) ));
    uint32_t *tmp = nullptr;
    cudaCheck(hipcub::DeviceScan::InclusiveSum(nullptr, tempScanStorageSize, tmp, tmp, MaxNumModules));
    cudaCheck(hipMalloc(&tempScanStorage_d, tempScanStorageSize));
  }

  SiPixelRawToClusterGPUKernel::~SiPixelRawToClusterGPUKernel() {
    // free device memory used for RawToDigi on GPU
    // free the GPU memory
    cudaCheck(hipFree(word_d));
    cudaCheck(hipFree(fedId_d));
    cudaCheck(hipFree(pdigi_d));
    cudaCheck(hipFree(xx_d));
    cudaCheck(hipFree(yy_d));
    cudaCheck(hipFree(adc_d));
    cudaCheck(hipFree(moduleInd_d));
    cudaCheck(hipFree(rawIdArr_d));
    cudaCheck(hipFree(error_d));
    cudaCheck(hipFree(data_d));

    // these are for the clusterizer
    cudaCheck(hipFree(moduleStart_d));
    cudaCheck(hipFree(clus_d));
    cudaCheck(hipFree(clusInModule_d));
    cudaCheck(hipFree(moduleId_d));
    cudaCheck(hipFree(gpuProduct_d));

    // originally from rechits
    cudaCheck(hipFree(tempScanStorage_d));
    cudaCheck(hipFree(clusModuleStart_d));
  }

  void SiPixelRawToClusterGPUKernel::initializeWordFed(int fedId, unsigned int wordCounterGPU, const cms_uint32_t *src, unsigned int length) {
    std::memcpy(word+wordCounterGPU, src, sizeof(cms_uint32_t)*length);
    std::memset(fedId_h+wordCounterGPU/2, fedId - 1200, length/2);
  }

  ////////////////////

  __device__ uint32_t getLink(uint32_t ww)  {
    return ((ww >> pixelgpudetails::LINK_shift) & pixelgpudetails::LINK_mask);
  }


  __device__ uint32_t getRoc(uint32_t ww) {
    return ((ww >> pixelgpudetails::ROC_shift ) & pixelgpudetails::ROC_mask);
  }


  __device__ uint32_t getADC(uint32_t ww) {
    return ((ww >> pixelgpudetails::ADC_shift) & pixelgpudetails::ADC_mask);
  }


  __device__ bool isBarrel(uint32_t rawId) {
    return (1==((rawId>>25)&0x7));
  }

  __device__ pixelgpudetails::DetIdGPU getRawId(const SiPixelFedCablingMapGPU * Map, uint32_t fed, uint32_t link, uint32_t roc) {
    uint32_t index = fed * MAX_LINK * MAX_ROC + (link-1) * MAX_ROC + roc;
    pixelgpudetails::DetIdGPU detId = { Map->RawId[index], Map->rocInDet[index], Map->moduleId[index] };
    return detId;
  }

  //reference http://cmsdoxygen.web.cern.ch/cmsdoxygen/CMSSW_9_2_0/doc/html/dd/d31/FrameConversion_8cc_source.html
  //http://cmslxr.fnal.gov/source/CondFormats/SiPixelObjects/src/PixelROC.cc?v=CMSSW_9_2_0#0071
  // Convert local pixel to pixelgpudetails::global pixel
  __device__ pixelgpudetails::Pixel frameConversion(bool bpix, int side, uint32_t layer, uint32_t rocIdInDetUnit, pixelgpudetails::Pixel local) {

    int slopeRow  = 0, slopeCol = 0;
    int rowOffset = 0, colOffset = 0;

    if (bpix) {

      if (side == -1 && layer != 1) { // -Z side: 4 non-flipped modules oriented like 'dddd', except Layer 1
        if (rocIdInDetUnit < 8) {
          slopeRow = 1;
          slopeCol = -1;
          rowOffset = 0;
          colOffset = (8-rocIdInDetUnit)*pixelgpudetails::numColsInRoc-1;
        }
        else {
          slopeRow  = -1;
          slopeCol  = 1;
          rowOffset = 2*pixelgpudetails::numRowsInRoc-1;
          colOffset = (rocIdInDetUnit-8)*pixelgpudetails::numColsInRoc;
        } // if roc
      }
      else { // +Z side: 4 non-flipped modules oriented like 'pppp', but all 8 in layer1
        if (rocIdInDetUnit < 8) {
          slopeRow  = -1;
          slopeCol  =  1;
          rowOffset = 2*pixelgpudetails::numRowsInRoc-1;
          colOffset = rocIdInDetUnit * pixelgpudetails::numColsInRoc;
        }
        else {
          slopeRow  = 1;
          slopeCol  = -1;
          rowOffset = 0;
          colOffset = (16-rocIdInDetUnit)*pixelgpudetails::numColsInRoc-1;
        }
      }

    }
    else { // fpix
      if (side==-1) { // pannel 1
        if (rocIdInDetUnit < 8) {
          slopeRow = 1;
          slopeCol = -1;
          rowOffset = 0;
          colOffset = (8-rocIdInDetUnit)*pixelgpudetails::numColsInRoc-1;
        }
        else {
          slopeRow = -1;
          slopeCol = 1;
          rowOffset = 2*pixelgpudetails::numRowsInRoc-1;
          colOffset = (rocIdInDetUnit-8)*pixelgpudetails::numColsInRoc;
        }
      }
      else { // pannel 2
        if (rocIdInDetUnit < 8) {
          slopeRow = 1;
          slopeCol = -1;
          rowOffset = 0;
          colOffset = (8-rocIdInDetUnit)*pixelgpudetails::numColsInRoc-1;
        }
        else {
          slopeRow = -1;
          slopeCol = 1;
          rowOffset = 2*pixelgpudetails::numRowsInRoc-1;
          colOffset = (rocIdInDetUnit-8)*pixelgpudetails::numColsInRoc;
        }

      } // side

    }

    uint32_t gRow = rowOffset+slopeRow*local.row;
    uint32_t gCol = colOffset+slopeCol*local.col;
    //printf("Inside frameConversion row: %u, column: %u\n",gRow, gCol);
    pixelgpudetails::Pixel global = {gRow, gCol};
    return global;
  }


  __device__ uint32_t conversionError(uint32_t fedId, uint32_t status, bool debug = false)
  {
    uint32_t errorType = 0;

    // debug = true;

    switch (status) {
      case(1) : {
        if (debug) printf("Error in Fed: %i, invalid channel Id (errorType = 35\n)", fedId );
        errorType = 35;
        break;
      }
      case(2) : {
        if (debug) printf("Error in Fed: %i, invalid ROC Id (errorType = 36)\n", fedId);
        errorType = 36;
        break;
      }
      case(3) : {
        if (debug) printf("Error in Fed: %i, invalid dcol/pixel value (errorType = 37)\n", fedId);
        errorType = 37;
        break;
      }
      case(4) : {
        if (debug) printf("Error in Fed: %i, dcol/pixel read out of order (errorType = 38)\n", fedId);
        errorType = 38;
        break;
      }
      default:
        if (debug) printf("Cabling check returned unexpected result, status = %i\n", status);
    };

    return errorType;
  }

  __device__ bool rocRowColIsValid(uint32_t rocRow, uint32_t rocCol)
  {
    uint32_t numRowsInRoc = 80;
    uint32_t numColsInRoc = 52;

    /// row and collumn in ROC representation
    return ((rocRow < numRowsInRoc) & (rocCol < numColsInRoc));
  }

  __device__ bool dcolIsValid(uint32_t dcol, uint32_t pxid)
  {
    return ((dcol < 26) &  (2 <= pxid) & (pxid < 162));
  }

  __device__ uint32_t checkROC(uint32_t errorWord, uint32_t fedId, uint32_t link, const SiPixelFedCablingMapGPU *Map, bool debug = false)
  {
    int errorType = (errorWord >> pixelgpudetails::ROC_shift) & pixelgpudetails::ERROR_mask;
    if (errorType < 25) return false;
    bool errorFound = false;

    switch (errorType) {
      case(25) : {
        errorFound = true;
        uint32_t index = fedId * MAX_LINK * MAX_ROC + (link-1) * MAX_ROC + 1;
        if (index > 1 && index <= Map->size) {
          if (!(link == Map->link[index] && 1 == Map->roc[index])) errorFound = false;
        }
        if (debug&errorFound) printf("Invalid ROC = 25 found (errorType = 25)\n");
        break;
      }
      case(26) : {
        if (debug) printf("Gap word found (errorType = 26)\n");
        errorFound = true;
        break;
      }
      case(27) : {
        if (debug) printf("Dummy word found (errorType = 27)\n");
        errorFound = true;
        break;
      }
      case(28) : {
        if (debug) printf("Error fifo nearly full (errorType = 28)\n");
        errorFound = true;
        break;
      }
      case(29) : {
        if (debug) printf("Timeout on a channel (errorType = 29)\n");
        if ((errorWord >> pixelgpudetails::OMIT_ERR_shift) & pixelgpudetails::OMIT_ERR_mask) {
          if (debug) printf("...first errorType=29 error, this gets masked out\n");
        }
        errorFound = true;
        break;
      }
      case(30) : {
        if (debug) printf("TBM error trailer (errorType = 30)\n");
        int StateMatch_bits = 4;
        int StateMatch_shift = 8;
        uint32_t StateMatch_mask = ~(~uint32_t(0) << StateMatch_bits);
        int StateMatch = (errorWord >> StateMatch_shift) & StateMatch_mask;
        if ( StateMatch != 1 && StateMatch != 8 ) {
          if (debug) printf("FED error 30 with unexpected State Bits (errorType = 30)\n");
        }
        if ( StateMatch == 1 ) errorType = 40; // 1=Overflow -> 40, 8=number of ROCs -> 30
        errorFound = true;
        break;
      }
      case(31) : {
        if (debug) printf("Event number error (errorType = 31)\n");
        errorFound = true;
        break;
      }
      default:
        errorFound = false;
    };

    return errorFound? errorType : 0;
  }

  __device__ uint32_t getErrRawID(uint32_t fedId, uint32_t errWord, uint32_t errorType, const SiPixelFedCablingMapGPU *Map, bool debug = false)
  {
    uint32_t rID = 0xffffffff;

    switch (errorType) {
      case 25 : case 30 : case 31 : case 36 : case 40 : {
        //set dummy values for cabling just to get detId from link
        //cabling.dcol = 0;
        //cabling.pxid = 2;
        uint32_t roc  = 1;
        uint32_t link = (errWord >> pixelgpudetails::LINK_shift) & pixelgpudetails::LINK_mask;
        uint32_t rID_temp = getRawId(Map, fedId, link, roc).RawId;
        if (rID_temp != 9999) rID = rID_temp;
        break;
      }
      case 29 : {
        int chanNmbr = 0;
        const int DB0_shift = 0;
        const int DB1_shift = DB0_shift + 1;
        const int DB2_shift = DB1_shift + 1;
        const int DB3_shift = DB2_shift + 1;
        const int DB4_shift = DB3_shift + 1;
        const uint32_t DataBit_mask = ~(~uint32_t(0) << 1);

        int CH1 = (errWord >> DB0_shift) & DataBit_mask;
        int CH2 = (errWord >> DB1_shift) & DataBit_mask;
        int CH3 = (errWord >> DB2_shift) & DataBit_mask;
        int CH4 = (errWord >> DB3_shift) & DataBit_mask;
        int CH5 = (errWord >> DB4_shift) & DataBit_mask;
        int BLOCK_bits      = 3;
        int BLOCK_shift     = 8;
        uint32_t BLOCK_mask = ~(~uint32_t(0) << BLOCK_bits);
        int BLOCK = (errWord >> BLOCK_shift) & BLOCK_mask;
        int localCH = 1*CH1+2*CH2+3*CH3+4*CH4+5*CH5;
        if (BLOCK%2==0) chanNmbr=(BLOCK/2)*9+localCH;
        else chanNmbr = ((BLOCK-1)/2)*9+4+localCH;
        if ((chanNmbr < 1)||(chanNmbr > 36)) break;  // signifies unexpected result

        // set dummy values for cabling just to get detId from link if in Barrel
        //cabling.dcol = 0;
        //cabling.pxid = 2;
        uint32_t roc  = 1;
        uint32_t link = chanNmbr;
        uint32_t rID_temp = getRawId(Map, fedId, link, roc).RawId;
        if(rID_temp != 9999) rID = rID_temp;
        break;
      }
      case 37 : case 38: {
        //cabling.dcol = 0;
        //cabling.pxid = 2;
        uint32_t roc  = (errWord >> pixelgpudetails::ROC_shift) & pixelgpudetails::ROC_mask;
        uint32_t link = (errWord >> pixelgpudetails::LINK_shift) & pixelgpudetails::LINK_mask;
        uint32_t rID_temp = getRawId(Map, fedId, link, roc).RawId;
        if(rID_temp != 9999) rID = rID_temp;
        break;
      }
      default:
        break;
    };

    return rID;
  }

  /*----------
   * Name: applyADCthreshold_kernel()
   * Desc: converts adc count to electrons and then applies the
   * threshold on each channel.
   * make pixel to 0 if it is below the threshold
   * Input: xx_d[], yy_d[], layer_d[], wordCounter, adc[], ADCThreshold
   *-----------
   * Output: xx_adc[], yy_adc[] with pixel threshold applied
   */
  // kernel to apply adc threshold on the channels


  // Felice: gains and pedestals are not the same for each pixel. This code should be rewritten to take
  // in account local gains/pedestals
  // __global__ void applyADCthreshold_kernel(const uint32_t *xx_d, const uint32_t *yy_d, const uint32_t *layer_d, uint32_t *adc, const uint32_t wordCounter,
  //  const ADCThreshold adcThreshold, uint32_t *xx_adc, uint32_t *yy_adc ) {
  //   int tid = threadIdx.x;
  //   int gIndex = blockDim.x*blockIdx.x+tid;
  //   if (gIndex<wordCounter) {
  //     uint32_t adcOld = adc[gIndex];
  //     const float gain = adcThreshold.theElectronPerADCGain_; // default: 1 ADC = 135 electrons
  //     const float pedestal = 0; //
  //     int adcNew = int(adcOld*gain+pedestal);
  //     // rare chance of entering into the if ()
  //     if (layer_d[gIndex]>=adcThreshold.theFirstStack_) {
  //       if (adcThreshold.theStackADC_==1 && adcOld==1) {
  //         adcNew = int(255*135); // Arbitrarily use overflow value.
  //       }
  //       if (adcThreshold.theStackADC_ >1 && adcThreshold.theStackADC_!=255 && adcOld>=1){
  //         adcNew = int((adcOld-1) * gain * 255/float(adcThreshold.theStackADC_-1));
  //       }
  //     }
  //
  //     if (adcNew >adcThreshold.thePixelThreshold ) {
  //       xx_adc[gIndex]=xx_d[gIndex];
  //       yy_adc[gIndex]=yy_d[gIndex];
  //     }
  //     else {
  //       xx_adc[gIndex]=0; // 0: dead pixel
  //       yy_adc[gIndex]=0;
  //     }
  //     adc[gIndex] = adcNew;
  //   }
  // }


  // Kernel to perform Raw to Digi conversion
  __global__ void RawToDigi_kernel(const SiPixelFedCablingMapGPU *Map, const unsigned char *modToUnp,
      const uint32_t wordCounter, const uint32_t *Word, const uint8_t *fedIds,
      uint16_t * XX, uint16_t * YY, uint16_t * ADC,
      uint32_t * pdigi, uint32_t *rawIdArr, uint16_t * moduleId,
      GPU::SimpleVector<pixelgpudetails::error_obj> *err,
      bool useQualityInfo, bool includeErrors, bool debug)
  {
    uint32_t blockId  = blockIdx.x;
    uint32_t threadId  = threadIdx.x;

    bool skipROC = false;
    //if (threadId==0) printf("Event: %u blockId: %u start: %u end: %u\n", eventno, blockId, begin, end);

    for (int aaa=0; aaa<1; ++aaa) {  // too many coninue below.... (to be fixed)
      auto gIndex = threadId + blockId*blockDim.x;
      if (gIndex < wordCounter) {

        uint32_t fedId = fedIds[gIndex/2]; // +1200;

        // initialize (too many coninue below)
        pdigi[gIndex]  = 0;
        rawIdArr[gIndex] = 0;
        moduleId[gIndex] = 9999;

        uint32_t ww = Word[gIndex]; // Array containing 32 bit raw data
        if (ww == 0) {
          //noise and dead channels are ignored
          XX[gIndex]    = 0;  // 0 is an indicator of a noise/dead channel
          YY[gIndex]    = 0; // skip these pixels during clusterization
          ADC[gIndex]   = 0;
          continue; // 0: bad word
        }

        uint32_t link  = getLink(ww);            // Extract link
        uint32_t roc   = getRoc(ww);             // Extract Roc in link
        pixelgpudetails::DetIdGPU detId = getRawId(Map, fedId, link, roc);

        uint32_t errorType = checkROC(ww, fedId, link, Map, debug);
        skipROC = (roc < pixelgpudetails::maxROCIndex) ? false : (errorType != 0);
        if (includeErrors and skipROC)
        {
          uint32_t rID = getErrRawID(fedId, ww, errorType, Map, debug);
          err->emplace_back(rID, ww, errorType, fedId);
          continue;
        }

        uint32_t rawId  = detId.RawId;
        uint32_t rocIdInDetUnit = detId.rocInDet;
        bool barrel = isBarrel(rawId);

        uint32_t index = fedId * MAX_LINK * MAX_ROC + (link-1) * MAX_ROC + roc;
        if (useQualityInfo) {

          skipROC = Map->badRocs[index];
          if (skipROC) continue;

        }
        skipROC = modToUnp[index];
        if (skipROC) continue;

        uint32_t layer = 0;//, ladder =0;
        int side = 0, panel = 0, module = 0;//disk = 0,blade = 0

        if (barrel)
        {
          layer  = (rawId >> pixelgpudetails::layerStartBit) & pixelgpudetails::layerMask;
          module = (rawId >> pixelgpudetails::moduleStartBit) & pixelgpudetails::moduleMask;
          side   = (module < 5)? -1 : 1;
        }
        else {
          // endcap ids
          layer = 0;
          panel = (rawId >> pixelgpudetails::panelStartBit) & pixelgpudetails::panelMask;
          //disk  = (rawId >> diskStartBit_) & diskMask_;
          side  = (panel == 1)? -1 : 1;
          //blade = (rawId >> bladeStartBit_) & bladeMask_;
        }

        // ***special case of layer to 1 be handled here
        pixelgpudetails::Pixel localPix;
        if (layer == 1) {
          uint32_t col = (ww >> pixelgpudetails::COL_shift) & pixelgpudetails::COL_mask;
          uint32_t row = (ww >> pixelgpudetails::ROW_shift) & pixelgpudetails::ROW_mask;
          localPix.row = row;
          localPix.col = col;
          if (includeErrors) {
            if (not rocRowColIsValid(row, col)) {
              uint32_t error = conversionError(fedId, 3, debug); //use the device function and fill the arrays
              err->emplace_back(rawId, ww, error, fedId);
              if(debug) printf("BPIX1  Error status: %i\n", error);
              continue;
            }
          }
        } else {
          // ***conversion rules for dcol and pxid
          uint32_t dcol = (ww >> pixelgpudetails::DCOL_shift) & pixelgpudetails::DCOL_mask;
          uint32_t pxid = (ww >> pixelgpudetails::PXID_shift) & pixelgpudetails::PXID_mask;
          uint32_t row  = pixelgpudetails::numRowsInRoc - pxid/2;
          uint32_t col  = dcol*2 + pxid%2;
          localPix.row = row;
          localPix.col = col;
          if (includeErrors and not dcolIsValid(dcol, pxid)) {
            uint32_t error = conversionError(fedId, 3, debug);
            err->emplace_back(rawId, ww, error, fedId);
            if(debug) printf("Error status: %i %d %d %d %d\n", error, dcol, pxid, fedId, roc);
            continue;
          }
        }

        pixelgpudetails::Pixel globalPix = frameConversion(barrel, side, layer, rocIdInDetUnit, localPix);
        XX[gIndex]    = globalPix.row;  // origin shifting by 1 0-159
        YY[gIndex]    = globalPix.col;  // origin shifting by 1 0-415
        ADC[gIndex]   = getADC(ww);
        pdigi[gIndex] = pixelgpudetails::pack(globalPix.row,globalPix.col,ADC[gIndex]);
        moduleId[gIndex] = detId.moduleId;
        rawIdArr[gIndex] = rawId;
      } // end of if (gIndex < end)
    } // end fake loop
  } // end of Raw to Digi kernel

  // Interface to outside
  void SiPixelRawToClusterGPUKernel::makeClustersAsync(
      const SiPixelFedCablingMapGPU *cablingMap,
      const unsigned char *modToUnp,
      const SiPixelGainForHLTonGPU *gains,
      const uint32_t wordCounter, const uint32_t fedCounter,
      bool convertADCtoElectrons,
      bool useQualityInfo, bool includeErrors, bool debug,
      cuda::stream_t<>& stream)
  {
    nDigis = wordCounter;

    const int threadsPerBlock = 512;
    const int blocks = (wordCounter + threadsPerBlock-1) /threadsPerBlock; // fill it all

    assert(0 == wordCounter%2);
    // wordCounter is the total no of words in each event to be trasfered on device
    cudaCheck(hipMemcpyAsync(&word_d[0],     &word[0],     wordCounter*sizeof(uint32_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(&fedId_d[0], &fedId_h[0], wordCounter*sizeof(uint8_t)/2, hipMemcpyDefault, stream.id()));

    constexpr uint32_t vsize = sizeof(GPU::SimpleVector<pixelgpudetails::error_obj>);
    constexpr uint32_t esize = sizeof(pixelgpudetails::error_obj);
    cudaCheck(hipMemcpyAsync(error_d, error_h_tmp, vsize, hipMemcpyDefault, stream.id()));

    // Launch rawToDigi kernel
    RawToDigi_kernel<<<blocks, threadsPerBlock, 0, stream.id()>>>(
        cablingMap,
        modToUnp,
        wordCounter,
        word_d,
        fedId_d,
        xx_d, yy_d, adc_d,
        pdigi_d,
        rawIdArr_d,
        moduleInd_d,
        error_d,
        useQualityInfo,
        includeErrors,
        debug);
    cudaCheck(hipGetLastError());

    // copy data to host variable

    cudaCheck(hipMemcpyAsync(pdigi_h, pdigi_d, wordCounter*sizeof(uint32_t), hipMemcpyDefault, stream.id()));
    cudaCheck(hipMemcpyAsync(rawIdArr_h, rawIdArr_d, wordCounter*sizeof(uint32_t), hipMemcpyDefault, stream.id()));

    if (includeErrors) {
      cudaCheck(hipMemcpyAsync(error_h, error_d, vsize, hipMemcpyDefault, stream.id()));
      cudaCheck(hipMemcpyAsync(data_h, data_d, MAX_FED*pixelgpudetails::MAX_WORD*esize, hipMemcpyDefault, stream.id()));
      // If we want to transfer only the minimal amount of data, we
      // need a synchronization point. A single ExternalWork (of
      // SiPixelRawToClusterHeterogeneous) does not help because it is
      // already used to synchronize the data movement. So we'd need
      // two ExternalWorks (or explicit use of TBB tasks). The
      // prototype of #100 would allow this easily (as there would be
      // two ExternalWorks).
      //
      //error_h->set_data(data_h);
      //cudaCheck(hipStreamSynchronize(stream.id()));
      //int size = error_h->size();
      //cudaCheck(hipMemcpyAsync(data_h, data_d, size*esize, hipMemcpyDefault, stream.id()));
    }
    // End  of Raw2Digi and passing data for cluserisation

    {
      // clusterizer ...
      using namespace gpuClustering;
      int threadsPerBlock = 256;
      int blocks = (wordCounter + threadsPerBlock - 1) / threadsPerBlock;

      gpuCalibPixel::calibDigis<<<blocks, threadsPerBlock, 0, stream.id()>>>(
          moduleInd_d,
          xx_d, yy_d, adc_d,
          gains,
          wordCounter);
      cudaCheck(hipGetLastError());

      // calibrated adc
      cudaCheck(hipMemcpyAsync(adc_h, adc_d, wordCounter*sizeof(uint16_t), hipMemcpyDefault, stream.id()));

      /*
         std::cout
         << "CUDA countModules kernel launch with " << blocks
         << " blocks of " << threadsPerBlock << " threads\n";
       */

      cudaCheck(hipMemsetAsync(moduleStart_d, 0x00, sizeof(uint32_t), stream.id()));

      countModules<<<blocks, threadsPerBlock, 0, stream.id()>>>(moduleInd_d, moduleStart_d, clus_d, wordCounter);
      cudaCheck(hipGetLastError());

      // read the number of modules into a data member, used by getProduct())
      cudaCheck(hipMemcpyAsync(&nModulesActive, moduleStart_d, sizeof(uint32_t), hipMemcpyDefault, stream.id()));

      threadsPerBlock = 256;
      blocks = MaxNumModules;
      /*
         std::cout << "CUDA findClus kernel launch with " << blocks
         << " blocks of " << threadsPerBlock << " threads\n";
       */
      cudaCheck(hipMemsetAsync(clusInModule_d, 0, (MaxNumModules)*sizeof(uint32_t), stream.id()));
      findClus<<<blocks, threadsPerBlock, 0, stream.id()>>>(
          moduleInd_d,
          xx_d, yy_d,
          moduleStart_d,
          clusInModule_d, moduleId_d,
          clus_d,
          wordCounter);
      cudaCheck(hipGetLastError());

      // count the module start indices already here (instead of
      // rechits) so that the number of clusters/hits can be made
      // available in the rechit producer without additional points of
      // synchronization/ExternalWork
      //
      // Set first the first element to 0
      cudaCheck(hipMemsetAsync(clusModuleStart_d, 0, sizeof(uint32_t), stream.id()));
      // Then use inclusive_scan to get the partial sum to the rest
      cudaCheck(hipcub::DeviceScan::InclusiveSum(tempScanStorage_d, tempScanStorageSize,
                                              clusInModule_d, &clusModuleStart_d[1], gpuClustering::MaxNumModules,
                                              stream.id()));
      // last element holds the number of all clusters
      cudaCheck(hipMemcpyAsync(&nClusters, clusModuleStart_d+gpuClustering::MaxNumModules, sizeof(uint32_t), hipMemcpyDefault, stream.id()));


      // clusters
      cudaCheck(hipMemcpyAsync(clus_h, clus_d, wordCounter*sizeof(uint32_t), hipMemcpyDefault, stream.id()));
    } // end clusterizer scope
  }

}
